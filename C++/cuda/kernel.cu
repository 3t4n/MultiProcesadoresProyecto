#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include "md5.h"
#include "utils.h"
//630188182
#define SIZEN 630188182 
#define MAX_LENGTH 7 
#define CHARSETLEN 26 


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


__device__ __constant__ unsigned char cudaCharSet[95];
__device__ unsigned char correctPass[MAX_LENGTH];

__global__ void bruteForce(unsigned char* prefixes, int length, int offset_start, int offeset_end, int prefix_to_calc, uint v1, uint v2, uint v3, uint v4, unsigned char* password_d)
{
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int local_threadId = threadIdx.x + threadIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
        + (threadIdx.y * blockDim.x) + threadIdx.x;
    //prefixes[offset + threadId + length - 1] = 'a';

    if (blockId < prefix_to_calc - 1 && local_threadId < CHARSETLEN - 1 && password_d[0]== '\0') {
        //Need to optimize 7758650592 
        memcpy(prefixes + (offeset_end + threadId) * MAX_LENGTH, prefixes + (offset_start + blockId) * MAX_LENGTH, (length - 1) * sizeof(char));
        prefixes[(offeset_end + threadId) * MAX_LENGTH + length - 1] = cudaCharSet[local_threadId];
        uint c1 = 0, c2 = 0, c3 = 0, c4 = 0;
        md5_vfy(prefixes + (offeset_end + threadId) * MAX_LENGTH, length, &c1, &c2, &c3, &c4);
        if (c1 == v1 && c2 == v2 && c3 == v3 && c4 == v4)
        {
            memcpy(password_d, prefixes + (offeset_end + threadId) * MAX_LENGTH, length * sizeof(char));
        }
    }
}

int main(int argc, unsigned char* argv[])
{
    char charset[] = "abcdefghijklmnopqrstuvwxyz";
    //af94ffbb0e815172b1160d4b58a3ece3 -> imanol
    //172346606e1d24062e891d537e917a90 -> lolol 
    unsigned char hash[33] = "eba021d91b44a97dec2588bbea58a447";
    unsigned char password[MAX_LENGTH], * password_d;


    uint v1, v2, v3, v4;
    md5_to_ints(hash, &v1, &v2, &v3, &v4);
    //  printf("%i %i %i %i", v1, v2, v3, v4);
    hipMemcpyToSymbol(HIP_SYMBOL(cudaCharSet), &charset, CHARSETLEN, 0, hipMemcpyHostToDevice);

    double ms;
    printf("Starting...\n");
    ms = 0;
    for (int h = 0; h < N; h++) {
        long prefix_n = 26;
        int length = 5;
        long offset_start = 0;
        long offest_end = 26;
        unsigned char* prefixes_d, * prefixes;
        prefixes = (unsigned char*)malloc(SIZEN * sizeof(char) * MAX_LENGTH);
        memset(prefixes, '\0', SIZEN * sizeof(unsigned char) * MAX_LENGTH);
        memset(password, '\0', sizeof(unsigned char) * MAX_LENGTH);

        //load first prefix
        for (int i = 0; i < CHARSETLEN; i++) prefixes[i * MAX_LENGTH] = charset[i];
        hipMalloc((void**)&prefixes_d, SIZEN * sizeof(unsigned char) * MAX_LENGTH);
        hipMalloc((void**)&password_d, MAX_LENGTH * sizeof(unsigned char));

        hipMemcpy(prefixes_d, prefixes, CHARSETLEN * MAX_LENGTH * sizeof(unsigned char), hipMemcpyHostToDevice);
        hipMemcpy(password_d, password, MAX_LENGTH * sizeof(unsigned char), hipMemcpyHostToDevice);

        start_timer();
        for (int i = 2; i <= length; i++) {
            long dim = (int)ceil(sqrt((float)prefix_n));
            dim3 grid_dim(dim, dim);
            prefix_n = prefix_n * CHARSETLEN;
            bruteForce << < grid_dim, 26 >> > (prefixes_d, i, offset_start, offest_end, prefix_n, v1, v2, v3, v4, password_d);
            hipDeviceSynchronize();
            gpuErrchk(hipPeekAtLastError());
            gpuErrchk(hipDeviceSynchronize());
            hipMemcpy(password, password_d, MAX_LENGTH * sizeof(unsigned char), hipMemcpyDeviceToHost);
            if (password[0] != '\0')
            {
                printf("We found %s as %s  ", hash, password);
                break;
            }

            offset_start = offest_end;
            offest_end = offset_start + prefix_n;

        }
        ms += stop_timer();
        free(prefixes);
        hipFree(prefixes_d);
        hipFree(password_d);
    }
    printf("avg time = %.5lf ms\n", (ms / N));


}